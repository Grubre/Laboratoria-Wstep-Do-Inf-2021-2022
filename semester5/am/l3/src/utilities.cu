#include "hip/hip_runtime.h"
#include "utilities.hpp"
#include <array>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <unordered_map>

__global__ void sort_by_fitness(int64_t *sorted_paths_indexes, int64_t *path_lengths, unsigned int node_count) {
    const auto island_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (island_id >= island_count) {
        return;
    }

    const int start_idx = island_id * island_size;
    const int end_idx = start_idx + island_size;

    // Initializing the indexes
    for (int i = start_idx; i < end_idx; i++) {
        sorted_paths_indexes[i] = i;
    }

    thrust::sort_by_key(thrust::device, path_lengths + start_idx, path_lengths + end_idx,
                        sorted_paths_indexes + start_idx, thrust::less<int64_t>());
}

__global__ void select(int64_t *paths, int64_t *temp, int64_t *sorted_paths_indexes, unsigned int node_count) {
    const auto path_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_id >= population_size) {
        return;
    }

    const auto offset_inside_island = path_id % island_size;

    auto selected_path_id = sorted_paths_indexes[path_id - p];
    if (offset_inside_island < p) {
        selected_path_id = sorted_paths_indexes[path_id];
    }

    const auto current_id = path_id;

    // copy paths[selected_path_id] to temp[path_id]
    for (auto i = 0u; i < node_count; i++) {
        temp[current_id * node_count + i] = paths[selected_path_id * node_count + i];
    }
}

void calculate_dist_between_nodes(int64_t *dist_between_nodes, const std::span<const Vec2> node_coords) {
    for (auto i = 0u; i < node_coords.size(); i++) {
        for (auto j = 0u; j < node_coords.size(); j++) {
            if (i == j)
                continue;
            dist_between_nodes[i + j * node_coords.size()] = dist(node_coords[i], node_coords[j]);
        }
    }
}

__global__ void calculate_path_lengths(int64_t *path_lengths, int64_t *paths, int64_t *dist_between_nodes,
                                       unsigned int population_size, unsigned int node_count) {
    const auto starting_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (starting_id >= population_size) {
        return;
    }

    // printf("calculating: %d\n", starting_id);

    int64_t path_length = 0;
    for (int i = 0; i < node_count - 1; i++) {
        const auto from = paths[node_count * starting_id + i];
        const auto to = paths[node_count * starting_id + i + 1];

        path_length += dist_between_nodes[from * node_count + to];
    }

    const auto last_node = paths[node_count * starting_id + node_count - 1];
    const auto first_node = paths[node_count * starting_id + 0];
    path_length += dist_between_nodes[last_node * node_count + first_node];

    // printf("starting_id = %d, path_length=%d\n", (int)starting_id,
    // (int)path_length);
    path_lengths[starting_id] = path_length;
}

__global__ void shuffle(int64_t *path, unsigned int node_count) {
    const auto starting_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (starting_id >= population_size) {
        return;
    }

    for (int i = 0; i < node_count; i++) {
        path[starting_id * node_count + i] = i;
    }

    hiprandState state;
    hiprand_init(starting_id + clock(), 0, 0,
                &state); // Unique seed for each thread
    for (int i = node_count - 1; i > 0; i--) {
        int j = hiprand(&state) % (i + 1);

        int64_t temp = path[starting_id * node_count + i];
        path[starting_id * node_count + i] = path[starting_id * node_count + j];
        path[starting_id * node_count + j] = temp;
    }
}

__global__ void get_best(int64_t *best_permutation_host, int64_t *best_length_host, int64_t *path_lengths,
                         int64_t *sorted_paths_indexes, int64_t *paths, unsigned int population_size,
                         unsigned int node_count) {
    uint64_t best_permutation_index = 0u;
    for (auto i = 0u; i < population_size; i++) {
        if (path_lengths[i] < path_lengths[best_permutation_index]) {
            best_permutation_index = i;
        }
    }

    if (*best_length_host <= path_lengths[best_permutation_index]) {
        return;
    }

    for (auto i = 0u; i < node_count; i++) {
        // printf("%d ", (int)paths[best_permutation_index * node_count + i]);
        best_permutation_host[i] = paths[best_permutation_index * node_count + i];
    }

    *best_length_host = path_lengths[best_permutation_index];
}

__global__ void mutate(int64_t *paths, unsigned int node_count, unsigned int number_of_mutations) {
    const auto path_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_id >= population_size) {
        return;
    }

    const auto offset_inside_island = path_id % island_size;
    if (offset_inside_island >= p) {
        return;
    }

    const auto island_id = path_id / island_size;
    const auto mutation_chance = 1.0 - (double)island_id / (double)island_count;

    hiprandState state;
    hiprand_init(path_id + clock(), 0, 0, &state);

    // random double between 0 and 1
    const auto random_double = hiprand_uniform_double(&state) - 0.5;

    if (random_double > mutation_chance) {
        return;
    }

    for (auto i = 0u; i < number_of_mutations; i++) {
        const auto random_node_a = hiprand(&state) % node_count;
        const auto random_node_b = hiprand(&state) % node_count;

        const auto temp = paths[path_id * node_count + random_node_a];
        paths[path_id * node_count + random_node_a] = paths[path_id * node_count + random_node_b];
        paths[path_id * node_count + random_node_b] = temp;
    }
}
