#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "common.hpp"
#include <iostream>
#include <array>
#include <span>
#include <unordered_map>
#include <stdio.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

constexpr auto population_size = 1000;

constexpr auto island_count = 1;
constexpr auto island_size = population_size / island_count;

constexpr auto selected_threshold = 0.8;

constexpr auto threadsPerBlock = 256;
constexpr auto blocksPerGrid = (population_size + threadsPerBlock - 1) / threadsPerBlock;

void calculate_dist_between_nodes(int64_t *dist_between_nodes, const std::span<const Vec2> node_coords) {
    for(auto i = 0u; i < node_coords.size(); i++) {
        for(auto j = 0u; j < node_coords.size(); j++) {
            if (i == j)
                continue;
            dist_between_nodes[i + j * node_coords.size()] = dist(node_coords[i], node_coords[j]);
        }
    }
}

__global__ void calculate_path_lengths(int64_t* path_lengths, int64_t *paths, int64_t *dist_between_nodes, unsigned int population_size, unsigned int node_count) {
    const auto starting_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (starting_id >= population_size) {
        return;
    }

    int64_t path_length = 0;
    for (int i = 0; i < node_count - 1; i++) {
        const auto from = paths[node_count * starting_id + i];
        const auto to = paths[node_count * starting_id + i + 1];

        path_length += dist_between_nodes[from * node_count + to];

    }

    const auto last_node = paths[node_count * starting_id + node_count - 1];
    const auto first_node = paths[node_count * starting_id + 0];
    path_length += dist_between_nodes[last_node * node_count + first_node];

    // printf("starting_id = %d, path_length=%d\n", (int)starting_id, (int)path_length);
    path_lengths[starting_id] = path_length;
}

__global__ void shuffle(int64_t* path, unsigned int node_count) {
    const auto starting_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (starting_id >= population_size) {
        return;
    }

    for (int i = 0; i < node_count; i++) {
        path[starting_id * node_count + i] = i;
    }

    hiprandState state;
    hiprand_init(starting_id + clock(), 0, 0, &state); // Unique seed for each thread
    for (int i = node_count - 1; i > 0; i--) {
        int j = hiprand(&state) % (i + 1);

        int64_t temp = path[starting_id * node_count + i];
        path[starting_id * node_count + i] = path[starting_id * node_count + j];
        path[starting_id * node_count + j] = temp;
    }
}

__global__ void get_best(int64_t* best_permutation_host, int64_t* best_length_host, int64_t* path_lengths, int64_t* paths, unsigned int population_size, unsigned int node_count) {
    uint64_t best_permutation_index = 0u;
    for(auto i = 0u; i < population_size; i++) {
        if (path_lengths[i] < path_lengths[best_permutation_index]) {
            best_permutation_index = i;
        }
    }

    for(auto i = 0u; i < node_count; i++) {
        best_permutation_host[i] = paths[best_permutation_index * node_count + i];
    }

    *best_length_host = path_lengths[best_permutation_index];
}

__global__ void sort_by_fitness(int64_t* sorted_paths_indexes, int64_t* path_lengths, unsigned int node_count) {
    const auto island_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (island_id >= island_count) {
        return;
    }

    const int start_idx = island_id * island_size;
    const int end_idx = start_idx + island_size;

    // Initializing the indexes
    for(int i = start_idx; i < end_idx; i++) {
        sorted_paths_indexes[i] = i;
    }

    thrust::sort_by_key(thrust::device, path_lengths + start_idx, path_lengths + end_idx, sorted_paths_indexes + start_idx, thrust::less<int64_t>());
}

__global__ void select(int64_t* paths, int64_t* sorted_paths_indexes, unsigned int node_count) {
    const auto path_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(path_id >= population_size) {
        return;
    }

    const auto island_id = path_id / island_size;
    const auto island_start_idx = island_id * island_size;
    const auto offset_inside_island = path_id % island_size;

    const auto p = (int)(island_size * selected_threshold);

    if(offset_inside_island < p) {
        return;
    }

    const auto selected_path_id = sorted_paths_indexes[path_id - p];
    const auto current_id = sorted_paths_indexes[path_id];

    // copy paths[selected_path_id] to paths[path_id]
    for(auto i = 0u; i < node_count; i++) {
        paths[current_id * node_count + i] = paths[selected_path_id * node_count + i];
    }
}

__global__ void crossover(int64_t* paths, int64_t* sorted_paths_indexes, unsigned int node_count) {
    const auto path_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(path_id >= population_size / 2) {
        return;
    }

    const auto parent_a = sorted_paths_indexes[2 * path_id];
    const auto parent_b = sorted_paths_indexes[2 * path_id + 1];


}

auto main(int argc, char** argv) -> int {
    if(argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <input_file> " << std::endl;
        return 1;
    }
    const auto nodes = read_file(argv[1]);

    // const auto nodes = std::vector<Vec2>{Vec2{0,0}, Vec2{0,1}, Vec2{0,2}, Vec2{0,3}, Vec2{0,4}};

    const auto node_count = nodes.size();

    // Calculate distances between each node
    auto dist_between_nodes_host = (int64_t*)malloc(node_count * node_count * sizeof(int64_t));
    calculate_dist_between_nodes(dist_between_nodes_host, std::span(nodes));

    // Allocate the array holding distances between nodes on the gpu and initialize
    int64_t *dist_between_nodes_device;
    hipMalloc((void**)&dist_between_nodes_device, node_count * node_count * sizeof(int64_t));
    hipMemcpy(dist_between_nodes_device, dist_between_nodes_host, node_count * node_count * sizeof(int64_t), hipMemcpyHostToDevice);

    // Allocate the array holding different paths(specimens) on the gpu and initialize (randomly shuffle each path)
    int64_t *paths_device;
    hipMalloc((void**)&paths_device, population_size * node_count * sizeof(int64_t));
    shuffle<<<blocksPerGrid, threadsPerBlock>>>(paths_device, node_count);

    // Allocate the array holding path lengths on the gpu and initialize
    int64_t *path_lengths_device;
    hipMalloc((void**)&path_lengths_device, population_size * sizeof(int64_t));
    calculate_path_lengths<<<blocksPerGrid, threadsPerBlock>>>(path_lengths_device, paths_device, dist_between_nodes_device, population_size, node_count);

    // Allocate the array that holds the sorted paths indexes and initialize
    int64_t* sorted_paths_indexes_device;
    hipMalloc((void**)&sorted_paths_indexes_device, population_size * sizeof(int64_t));

    // Allocate the array that holds the offsprings and initialize
    int64_t* offsprings_device;
    hipMalloc((void**)&offsprings_device, population_size * node_count * sizeof(int64_t));

    // Run the genetic algorithm
    constexpr auto generations = 1000;
    constexpr auto internal_generations = 10;
    for(auto i = 0u; i < generations; i++) {
        for(auto j = 0u; j < internal_generations; j++) {
            // selection
            sort_by_fitness<<<blocksPerGrid, threadsPerBlock>>>(sorted_paths_indexes_device, path_lengths_device, node_count);
            // NOTE: Before doing crossover and mutation, for population_size=5000 i was getting the shortest path ~300_000 which seems large
            select<<<blocksPerGrid, threadsPerBlock>>>(paths_device, sorted_paths_indexes_device, node_count);
            // crossover, one thread for each offsprings_device element
            // crossover<<<blockPerGrid, threadsPerBlock>>>(offsprings_device, paths_device, sorted_paths_indexes_device, node_count);
            calculate_path_lengths<<<blocksPerGrid, threadsPerBlock>>>(path_lengths_device, paths_device, dist_between_nodes_device, population_size, node_count);
        }
        hipDeviceSynchronize();
        std::cout << "\r" << i + 1 << "/" << generations << std::flush;
    }
    std::cout << std::endl;

    // sort_by_fitness<<<blocksPerGrid, threadsPerBlock>>>(sorted_paths_indexes_device, path_lengths_device, node_count);
    // select<<<blocksPerGrid, threadsPerBlock>>>(paths_device, sorted_paths_indexes_device, node_count);

    // Get the solution in unified memory
    int64_t* best_length_host;
    int64_t* best_permutation_host;
    hipMallocManaged(&best_permutation_host, node_count * sizeof(int64_t));
    hipMallocManaged(&best_length_host, sizeof(int64_t));
    get_best<<<1, 1>>>(best_permutation_host, best_length_host, path_lengths_device, paths_device, population_size, node_count);
    hipDeviceSynchronize();

    auto paths_host = (int64_t*)malloc(population_size * node_count * sizeof(int64_t));
    hipMemcpy(paths_host,paths_device, node_count*population_size*sizeof(int64_t),hipMemcpyDeviceToHost);

    auto path_lengths_host = (int64_t*)malloc(population_size * sizeof(int64_t));
    hipMemcpy(path_lengths_host,path_lengths_device, population_size*sizeof(int64_t),hipMemcpyDeviceToHost);

    auto sorted_paths_indexes_host = (int64_t*)malloc(population_size * sizeof(int64_t));
    hipMemcpy(sorted_paths_indexes_host,sorted_paths_indexes_device, population_size*sizeof(int64_t),hipMemcpyDeviceToHost);

    // for(auto i = 0u; i < population_size; i++) {
    //     std::cout << sorted_paths_indexes_host[i] << " ";
    //     printf("\n");
    // }
    // for(auto i = 0u; i < population_size; i++) {
    //     if(i%island_size==0) {
    //         std::cout << "\n======================" << std::endl;
    //     }
    //     for(int j = 0u; j < node_count; j++) {
    //         std::cout << paths_host[sorted_paths_indexes_host[i] * node_count + j] << " ";
    //         // printf("paths_host[%zu]=%d\n", i * node_count + j, (int)paths_host[i * node_count + j]);
    //     }
    //     std::cout << "\tlength = " << path_lengths_host[i] << std::endl;
    //     // std::cout << "======================" << std::endl;
    // }

    printf("%ld\n", * best_length_host);

    return 0;
}
