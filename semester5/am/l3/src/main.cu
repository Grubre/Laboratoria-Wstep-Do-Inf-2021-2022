#include "hip/hip_runtime.h"
#include "utilities.hpp"
#include <array>
#include <filesystem>
#include <iostream>
#include <span>
#include <stdio.h>
#include <unordered_map>

// taken from https://www.hindawi.com/journals/mpe/2020/1398595/
__device__ void ro(int64_t *a, int64_t *temp, int64_t len, int64_t num, unsigned int node_count) {
    int pos;
    for (int i = 0; i < len; i++) {
        if (a[i] == num) {
            pos = i;
        }
    }
    int j = 0;
    for (int i = pos; i < len; i++) {
        temp[j] = a[i];
        j++;
    }
    for (int k = 0; k < pos; k++) {
        temp[j] = a[k];
        j++;
    }
    for (int i = 0; i < len; i++) {
        a[i] = temp[i];
    }
}

// based on https://www.hindawi.com/journals/mpe/2020/1398595/
__global__ void crossover(int64_t *paths, int64_t *temp_buf, int64_t *dist_between_nodes, unsigned int node_count) {
    const auto pair_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (pair_id >= population_size / 2) {
        return;
    }

    const auto offset_inside_island = (2 * pair_id) % island_size;
    if (offset_inside_island >= p) {
        return;
    }

    hiprandState state;
    hiprand_init((unsigned long long)clock() + pair_id, 0, 0, &state);

    const auto parent_a_offset = 2 * pair_id;
    const auto parent_b_offset = 2 * pair_id + 1;
    const auto parent_a = paths + parent_a_offset * node_count;
    const auto parent_b = paths + parent_b_offset * node_count;

    // const auto trial = hiprand_uniform(&state);
    // if (trial > crossover_chance) {
    //     return;
    // }

    // printf("pair %d: a: %d, b: %d\n", pair_id, parent_a[0], parent_b[0]);
    const auto result = temp_buf + pair_id * node_count;
    const auto temp = temp_buf + (population_size / 2 + pair_id) * node_count;

    const auto start = hiprand(&state) % node_count;

    result[0] = start;

    ro(parent_a, temp, node_count, start, node_count);
    ro(parent_b, temp, node_count, start, node_count);

    for (auto i = 0u; i < node_count - 1; i++) {
        const auto from_a = parent_a[i];
        const auto from_b = parent_b[i];
        const auto to_a = parent_a[i + 1];
        const auto to_b = parent_b[i + 1];

        const auto dist_from_a = dist_between_nodes[from_a * node_count + to_a];
        const auto dist_from_b = dist_between_nodes[from_b * node_count + to_b];

        if (dist_from_a < dist_from_b) {
            result[i + 1] = to_a;
            ro(paths + parent_b_offset * node_count + i + 1, temp, node_count - i - 1, to_a, node_count);
        } else {
            result[i + 1] = to_b;
            ro(paths + parent_a_offset * node_count + i + 1, temp, node_count - i - 1, to_b, node_count);
        }
    }

    for (auto i = 0u; i < node_count; i++) {
        paths[parent_a_offset * node_count + i] = result[i];
    }

    const auto midpoint = node_count / 2;
    auto j = 0u;

    for (auto i = midpoint; i < node_count; i++) {
        paths[parent_b_offset * node_count + j] = result[i];
        j++;
    }

    for (auto i = 0u; i < midpoint; i++) {
        paths[parent_b_offset * node_count + j] = result[i];
        j++;
    }
}

void print_population(int64_t *paths_device, int64_t *paths_host, int64_t *path_lengths_host,
                      int64_t *path_lengths_device, unsigned int node_count) {
    hipMemcpy(paths_host, paths_device, node_count * population_size * sizeof(int64_t), hipMemcpyDeviceToHost);

    hipMemcpy(path_lengths_host, path_lengths_device, population_size * sizeof(int64_t), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    for (auto i = 0u; i < population_size; i++) {
        if (i % island_size == 0) {
            std::cout << "\n======================" << std::endl;
        }
        for (int j = 0u; j < node_count; j++) {
            // std::cout
            //     << paths_host[i * node_count + j]
            //     << " ";
            std::cout << paths_host[i * node_count + j] << " ";
            // printf("paths_host[%zu]=%d\n", i * node_count + j,
        }
        std::cout << "\tlength = " << path_lengths_host[i] << std::endl;
        // std::cout << "======================" << std::endl;
    }
}

__global__ void exchange(int64_t *paths, unsigned int node_count) {
    const auto path_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_id >= population_size) {
        return;
    }

    const auto island_id = path_id / island_size;
    const auto offset_inside_island = path_id % island_size;

    if (offset_inside_island >= exchange_cutoff_index) {
        return;
    }

    const auto destination_island_id = (island_id + 1) % island_count;
    const auto destination_path_id = destination_island_id * island_size + island_size - offset_inside_island - 1;

    for (auto i = 0u; i < node_count; i++) {
        paths[destination_path_id * node_count + i] = paths[path_id * node_count + i];
    }
}

auto main(int argc, char **argv) -> int {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <input_file> " << std::endl;
        return 1;
    }
    const auto nodes = read_file(argv[1]);

    // const auto nodes = std::vector<Vec2>{Vec2{0, 0},  Vec2{0, 1},  Vec2{0, 2},  Vec2{0, 3},  Vec2{0, 4},
    //                                      Vec2{0, 5},  Vec2{0, 6},  Vec2{0, 7},  Vec2{0, 8},  Vec2{0, 9},
    //                                      Vec2{0, 10}, Vec2{0, 11}, Vec2{0, 12}, Vec2{0, 13}, Vec2{0, 14}};

    const auto node_count = nodes.size();

    // Calculate distances between each node
    auto dist_between_nodes_host = (int64_t *)malloc(node_count * node_count * sizeof(int64_t));
    calculate_dist_between_nodes(dist_between_nodes_host, std::span(nodes));

    // Allocate the array holding distances between nodes on the gpu and
    // initialize
    int64_t *dist_between_nodes_device;
    hipMalloc((void **)&dist_between_nodes_device, node_count * node_count * sizeof(int64_t));
    hipMemcpy(dist_between_nodes_device, dist_between_nodes_host, node_count * node_count * sizeof(int64_t),
               hipMemcpyHostToDevice);

    // Allocate the array holding different paths(specimens) on the gpu and
    // initialize (randomly shuffle each path)
    int64_t *paths_device;
    hipMalloc((void **)&paths_device, population_size * node_count * sizeof(int64_t));
    shuffle<<<blocksPerGrid, threadsPerBlock>>>(paths_device, node_count);

    // Allocate the array holding path lengths on the gpu and initialize
    int64_t *path_lengths_device;
    hipMalloc((void **)&path_lengths_device, population_size * sizeof(int64_t));
    calculate_path_lengths<<<blocksPerGrid, threadsPerBlock>>>(path_lengths_device, paths_device,
                                                               dist_between_nodes_device, population_size, node_count);

    // Allocate the array that holds the sorted paths indexes and initialize
    int64_t *sorted_paths_indexes_device;
    hipMalloc((void **)&sorted_paths_indexes_device, population_size * sizeof(int64_t));

    // Allocate memory for the results on the host and in unified memory
    int64_t *best_length_host;
    int64_t *best_permutation_host;
    auto paths_host = (int64_t *)malloc(population_size * node_count * sizeof(int64_t));
    auto path_lengths_host = (int64_t *)malloc(population_size * sizeof(int64_t));
    hipMallocManaged(&best_permutation_host, node_count * sizeof(int64_t));
    hipMallocManaged(&best_length_host, sizeof(int64_t));

    *best_length_host = std::numeric_limits<int64_t>::max();

    // Run the genetic algorithm
    // constexpr auto generations = 1000;
    // constexpr auto internal_generations = 10;

    // NOTE: Before doing crossover and mutation, for
    // population_size=5000 i was getting the shortest path
    // ~300_000
    // which seems large

    int64_t *temp_device;
    hipMalloc((void **)&temp_device, population_size * node_count * sizeof(int64_t));

    sort_by_fitness<<<blocksPerGrid, threadsPerBlock>>>(sorted_paths_indexes_device, path_lengths_device, node_count);

    constexpr auto generations = 500;
    constexpr auto internal_generations = 10;
    for (auto i = 0u; i < generations; i++) {
        for (auto j = 0u; j < internal_generations; j++) {
            // selection
            select<<<blocksPerGrid, threadsPerBlock>>>(paths_device, temp_device, sorted_paths_indexes_device,
                                                       node_count);
            hipMemcpy(paths_device, temp_device, population_size * node_count * sizeof(int64_t),
                       hipMemcpyDeviceToDevice);
            crossover<<<blocksPerGrid, threadsPerBlock>>>(paths_device, temp_device, dist_between_nodes_device,
                                                          node_count);
            const auto number_of_mutations = i * node_count / (4 * generations);
            mutate<<<blocksPerGrid, threadsPerBlock>>>(paths_device, node_count, number_of_mutations);
            calculate_path_lengths<<<blocksPerGrid, threadsPerBlock>>>(
                path_lengths_device, paths_device, dist_between_nodes_device, population_size, node_count);
            sort_by_fitness<<<blocksPerGrid, threadsPerBlock>>>(sorted_paths_indexes_device, path_lengths_device,
                                                                node_count);
        }
        exchange<<<blocksPerGrid, threadsPerBlock>>>(paths_device, node_count);
        calculate_path_lengths<<<blocksPerGrid, threadsPerBlock>>>(
            path_lengths_device, paths_device, dist_between_nodes_device, population_size, node_count);
        sort_by_fitness<<<blocksPerGrid, threadsPerBlock>>>(sorted_paths_indexes_device, path_lengths_device,
                                                            node_count);
        calculate_path_lengths<<<blocksPerGrid, threadsPerBlock>>>(
            path_lengths_device, paths_device, dist_between_nodes_device, population_size, node_count);
        get_best<<<1, 1>>>(best_permutation_host, best_length_host, path_lengths_device, sorted_paths_indexes_device,
                           paths_device, population_size, node_count);
        hipDeviceSynchronize();
        std::cout << "\r" << i + 1 << "/" << generations << std::flush;
    }
    std::cout << std::endl;

    // sort_by_fitness<<<blocksPerGrid, threadsPerBlock>>>(
    //     sorted_paths_indexes_device, path_lengths_device, node_count);
    // select<<<blocksPerGrid, threadsPerBlock>>>(
    //     paths_device, temp_device, sorted_paths_indexes_device, node_count);
    // hipMemcpy(paths_device, temp_device,
    //            population_size * node_count * sizeof(int64_t),
    //            hipMemcpyDeviceToDevice);
    // // printf("After selection:\n");
    // // print_population(paths_device, paths_host, path_lengths_host,
    // //                  path_lengths_device, node_count);
    //
    // crossover<<<blocksPerGrid, threadsPerBlock>>>(
    //     paths_device, temp_device, dist_between_nodes_device, node_count);
    // mutate<<<blocksPerGrid, threadsPerBlock>>>(paths_device, node_count);
    // calculate_path_lengths<<<blocksPerGrid, threadsPerBlock>>>(
    //     path_lengths_device, paths_device, dist_between_nodes_device,
    //     population_size, node_count);
    //
    // printf("%ld\n", *best_length_host);
    // for (auto i = 0u; i < node_count; i++) {
    //     printf("%ld ", best_permutation_host[i]);
    // }
    //
    // auto length = 0llu;
    // for (auto i = 0u; i < node_count; i++) {
    //     length += dist_between_nodes_host[best_permutation_host[i] * node_count + best_permutation_host[i + 1]];
    // }
    // length += dist_between_nodes_host[best_permutation_host[node_count - 1] * node_count + best_permutation_host[0]];
    // printf("\ncalculated length %llu\n", length);


    const auto input_file = std::filesystem::path(argv[1]).filename().string();
    std::cout << "file: " << input_file << " -> best: " << *best_length_host << std::endl;
    auto output = std::ofstream("../output_" + input_file);

    output << *best_length_host << std::endl;

    for (auto i = 0u; i < node_count; i++) {
        output << nodes[best_permutation_host[i]].x << " " << nodes[best_permutation_host[i]].y << std::endl;
    }

    return 0;
}
