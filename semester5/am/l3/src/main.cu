#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "common.hpp"
#include <iostream>
#include <array>
#include <span>
#include <unordered_map>
#include <stdio.h>

constexpr auto population_size = 5;

constexpr auto threadsPerBlock = 256;
constexpr auto blocksPerGrid = (population_size + threadsPerBlock - 1) / threadsPerBlock;

void calculate_dist_between_nodes(int64_t *dist_between_nodes, const std::span<const Vec2> node_coords) {
    for(auto i = 0u; i < node_coords.size(); i++) {
        for(auto j = 0u; j < node_coords.size(); j++) {
            if (i == j)
                continue;
            dist_between_nodes[i + j * node_coords.size()] = dist(node_coords[i], node_coords[j]);
        }
    }
}

__global__ void calculate_path_lengths(int64_t* path_lengths, int64_t *paths, int64_t *dist_between_nodes, unsigned int population_size, unsigned int node_count) {
    const auto individual_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (individual_id >= population_size) {
        return;
    }

    int64_t path_length = 0;
    for (int i = 0; i < population_size - 1; i++) {
        // Assuming the individuals are stored in path_lengths in a flattened manner
        // and each individual is a sequence of node indices.
        const auto from = paths[population_size * individual_id + i];
        const auto to = paths[population_size * individual_id + i + 1];

        path_length += dist_between_nodes[from * node_count + to];
    }

    const auto last_node = paths[population_size * individual_id + population_size - 1];
    const auto first_node = paths[population_size * individual_id + 0];
    path_length += dist_between_nodes[last_node * node_count + first_node];

    // Write the total path length back
    path_lengths[individual_id] = path_length;
}

__global__ void shuffle(int64_t* path, unsigned int node_count) {
    unsigned int individual_id = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("individual_id=%d\n", individual_id);

    if (individual_id >= population_size) {
        return;
    }

    // Initialize the path for this individual
    for (int i = 0; i < node_count; i++) {
        path[individual_id * node_count + i] = i;
        // printf("path[%d]=%d\n", individual_id * node_count + i, (int)path[individual_id * node_count + i]);
    }

    // Shuffle the path using Fisher-Yates algorithm
    hiprandState state;
    hiprand_init(individual_id + clock(), 0, 0, &state); // Unique seed for each thread
    for (int i = node_count - 1; i > 0; i--) {
        int j = hiprand(&state) % (i + 1);

        // Swap i and j
        int64_t temp = path[individual_id * node_count + i];
        path[individual_id * node_count + i] = path[individual_id * node_count + j];
        path[individual_id * node_count + j] = temp;
    }
}

auto main(int argc, char** argv) -> int {
    if(argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <input_file> " << std::endl;
        return 1;
    }
    const auto nodes = read_file(argv[1]);

    // const auto nodes = std::vector<Vec2>{Vec2{0,0}, Vec2{0,1}, Vec2{0,2}, Vec2{0,3}, Vec2{0,4}};

    const auto node_count = nodes.size();

    // Calculate distances between each node
    auto dist_between_nodes_host = new int64_t[node_count * node_count];
    calculate_dist_between_nodes(dist_between_nodes_host, std::span(nodes));

    // Allocate the array holding distances between nodes on the gpu and initialize
    int64_t *dist_between_nodes_device;
    hipMalloc((void**)&dist_between_nodes_device, node_count * node_count * sizeof(int64_t));
    hipMemcpy(dist_between_nodes_device, dist_between_nodes_host, node_count * node_count * sizeof(int64_t), hipMemcpyHostToDevice);

    // Allocate the array holding different paths(specimens) on the gpu and initialize
    int64_t *paths_device;
    hipMalloc((void**)&paths_device, population_size * node_count * sizeof(int64_t));
    shuffle<<<blocksPerGrid, threadsPerBlock>>>(paths_device, node_count);

    // Allocate the array holding path lengths on the gpu and initialize
    int64_t *path_lengths_device;
    hipMalloc((void**)&path_lengths_device, population_size * sizeof(int64_t));
    calculate_path_lengths<<<blocksPerGrid, threadsPerBlock>>>(path_lengths_device, paths_device, dist_between_nodes_device, population_size, node_count);

    int64_t *paths_host = (int64_t*)malloc(population_size * node_count * sizeof(int64_t));
    hipMemcpy(paths_host,paths_device, node_count*population_size*sizeof(int64_t),hipMemcpyDeviceToHost);
    for(auto i = 0u; i < population_size; i++) {
        for(int j = 0u; j < node_count; j++) {
            std::cout << paths_host[i * node_count + j] << std::endl;
            // printf("paths_host[%zu]=%d\n", i * node_count + j, (int)paths_host[i * node_count + j]);
        }
        std::cout << "======================" << std::endl;
    }

    return 0;
}
